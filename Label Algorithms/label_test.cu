#include <iostream>
#include <string>
#include <sstream>
#include <hip/hip_runtime.h>
#include "ppm.h"
#include "labeling_cuda.cu"


#define START_TIME hipEventRecord(start,0)
#define STOP_TIME  hipEventRecord(stop,0 ); \
                   hipEventSynchronize(stop); \
                   hipEventElapsedTime( &et, start, stop )

void debug_writeLabels(int,int,int*,int);


int main(int argc, char* argv[]) {
	hipEvent_t start,stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	float et;
	int *label;

	for(int i=0; i<9; i++){
		std::stringstream fname;
		fname << "imagens/teste" << i << ".ppm";
		ppm image(fname.str());

		int w = image.width;
		int h = image.height;

		label = (int*)malloc(w*h*sizeof(int));

		START_TIME;
		CCL(image.r.data(), w, h, label);
		STOP_TIME;
		
		debug_writeLabels(w,h,label,i);

		std::cout << et << "  " ;

		free(label);
	}
	std::cout << std::endl;

	return 0;
}

void debug_writeLabels(int w, int h,int *label, int it){
	int *ptr = label;

	ppm lbl(w, h);

	for(int i = 0; i<h; i++){
		for(int j = 0; j<w; j++){
			lbl.r[i*w + j] = *ptr;
			ptr++;
		}
		std::cout << std::endl;
	}

	std::stringstream name;
	name << "imagens/cuda_output" << it << ".ppm";
	lbl.write(name.str());
}
